#include "hip/hip_runtime.h"
/*
    Merges the serial and cuda mockups to test if they give the same results on randomly generated test cases.
*/

#include <stdio.h>
#include "utils/io.c"
#include "utils/printer.c"
#include "serial_constraint.c"
//#include "domain_functions.c"
#include "utils/error_handler.cu"
#include "cuda_constraint.cu"
#include "utils/pcg-c-basic-0.9/pcg_basic.c"

void build_reverse_matrix(int,int*, int*);
int serial_constraint(int, int*, int*, uint32_t*, uint32_t*);
int cuda_constraint(int, int*, int*, uint32_t*, uint32_t*);
void build_reverse_matrix(int,int*, int*);
int* make_random_preference_matrix(int);
void make_full_domain(int,uint32_t*);
void make_partial_domain(int,uint32_t*);
void clone_domain(int, uint32_t*,uint32_t*);
void get_block_number_and_dimension(int, int, int*, int*);
int compare_domains(int, uint32_t*, uint32_t*);

/*
    Executes the tests.
    Use: takes as input the size of the test instances (the value of n), the number of tests to be executed with a complete domain,
    and the number of tests to be executed with randomly filled domains.
*/
int main(int argc, char *argv[]) {
    //get parameters from command line arguments
    int n, compl_tests, incompl_tests;
    int empty_domains_founds = 0;
    int errors = 0;
    int empty_notempty_errors = 0;
    int serial_status, cuda_status;
    if (argc == 4) {
        n = strtol(argv[1],NULL,10);
        compl_tests = strtol(argv[2],NULL,10);
        incompl_tests = strtol(argv[3],NULL,10);
    } else {
        printf("Process interrupted: wrong number of arguments.\nUse:\ncmp_constr <n> <number_of_tests_with_complete_domains> <number_of_tests_with_incomplete_domains>");
        return 5;
    }
    int total_tests = compl_tests + incompl_tests;

    //Initializes (seeds) global rng (see documentation at https://www.pcg-random.org/using-pcg-c-basic.html#pcg32-srandom-r-rngptr-initstate-initseq)
    pcg32_srandom(42, 42);

    int *men_pl, *women_pl;
    uint32_t *men_domain = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
    uint32_t *women_domain = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
    uint32_t *men_domain_parallel = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
    uint32_t *women_domain_parallel = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
    uint32_t *men_domain_orig = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
    uint32_t *women_domain_orig = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));

    for(int i=0; i<total_tests; i++){
        //printf("Beginning test %i\n",i);
        men_pl = make_random_preference_matrix(n);
        women_pl = make_random_preference_matrix(n);
        if(i<compl_tests){
            make_full_domain(n,men_domain);
            make_full_domain(n,women_domain);
        } else {
            make_partial_domain(n,men_domain);
            make_partial_domain(n,women_domain);
        }

        clone_domain(n,men_domain,men_domain_parallel);
        clone_domain(n,women_domain,women_domain_parallel);
        clone_domain(n,men_domain,men_domain_orig);
        clone_domain(n,women_domain,women_domain_orig);

        serial_status = serial_constraint(n,men_pl,women_pl,men_domain,women_domain);
        cuda_status = cuda_constraint(n,men_pl,women_pl,men_domain_parallel,women_domain_parallel);
        
        if (serial_status == -1 && serial_status == cuda_status){ //correct
            empty_domains_founds++;
        } else if(serial_status == cuda_status){ //must check
            if(!compare_domains(n,men_domain,men_domain_parallel) || !compare_domains(n,women_domain,women_domain_parallel)){//not equal
                errors++;
                printf("--------------------------------\n  Found error in test number %i (different domains) \n--------------------------------\n",i);
                print_preference_lists(n,men_pl,women_pl);
                print_domains(n,men_domain_orig,women_domain_orig);
                printf("Resulting domains for serial constraint:\n");
                print_domains(n,men_domain,women_domain);
                printf("Resulting domains for parallel constraint:\n");
                print_domains(n,men_domain_parallel,women_domain_parallel);
            }
        }else{ //surely an error
            errors++;
            empty_notempty_errors++;
            printf("--------------------------------\n  Found error in test number %i (empty and non empty domains) \n--------------------------------\n",i);
            if(serial_status == -1){
                printf("Domain for serial constraint was found to be empty.\n");
            } else {
                printf("Domain for parallel constraint was found to be empty.\n");
            }
            print_preference_lists(n,men_pl,women_pl);
            print_domains(n,men_domain_orig,women_domain_orig);
            printf("Resulting domains for serial constraint:\n");
            print_domains(n,men_domain,women_domain);
            printf("Resulting domains for parallel constraint:\n");
            print_domains(n,men_domain_parallel,women_domain_parallel);
        }
    }
    free(men_pl);
    free(women_pl);
    free(men_domain);
    free(women_domain);
    free(men_domain_parallel);
    free(women_domain_parallel);
    free(men_domain_orig);
    free(women_domain_orig);

    printf("\nTesting complete\n%i errors were found out of %i tests (of which, %i where one domain was empty and the other was not)\n%i empty domains were correctly identified",errors,total_tests,empty_notempty_errors,empty_domains_founds);
    return 0;
}


/*
    SERIAL CONSTRAINT
*/

int serial_constraint(int n, int *xpl, int *ypl, uint32_t *x_domain, uint32_t *y_domain) {

    //Returns if it finds an empty domain
    int emptyM, emptyW;
    for(int i=0;i<n;i++){
        emptyM = true;
        emptyW = true;
        for(int j=0;j<n;j++){
            if(getDomainBit(x_domain,i,j,n)){
                emptyM=false;
            }
            if(getDomainBit(y_domain,i,j,n)){
                emptyW=false;
            }
        }
        if(emptyM || emptyW){
            //printf("Found empty domain as input: returning.\n");
            return -1;
        }
    }

    //Builds the reverse matrixes
    int *xPy, *yPx;
    xPy = (int *)malloc(n * n * sizeof(int));
    yPx = (int *)malloc(n * n * sizeof(int));
    build_reverse_matrix(n,xpl,xPy);
    build_reverse_matrix(n,ypl,yPx);

    //print_reverse_matrixes(n,xPy,yPx);

    //Initializes xlb and yub
    int *xlb = (int *)malloc(n*sizeof(int));
    int *yub = (int *)malloc(n*sizeof(int));
    for(int i=0;i<n;i++){
        xlb[i]=0;
        yub[i]=n-1;
    }

    //applies once the constraint
    uint32_t *old_x_domain = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
    uint32_t *old_y_domain = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
    uint32_t *prev_x_domain = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
    uint32_t *prev_y_domain = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
    //printf("Before initialization of vectors.\n");
    for(int i=0;i<(n * n) / 32 + (n % 32 != 0);i++){
        old_x_domain[i]=4294967295;
        prev_x_domain[i]=4294967295;
        old_y_domain[i]=4294967295;
        prev_y_domain[i]=4294967295;
    }
    //printf("After initialization of vectors.\n");

    init(n,x_domain,y_domain,xpl,ypl,xPy,yPx,xlb);
    //printf("After init.\n");

    int stop;
    while(1){
        stop=1;
        for(int i=0;i<n;i++){
            //printf("%iith iteration\n",i);
            if(getMin(n,x_domain,i)>=n||getMax(n,y_domain,i)<0){
                //printf("\n-------------------\nFound empty domain!\n-------------------\n");
                //print_domains(n,x_domain,y_domain);
                return -1;
            }
            if(getMin(n,x_domain,i)!=getMin(n,old_x_domain,i)){
                deltaMin(i,n,x_domain,y_domain,xpl,ypl,xPy,yPx,xlb);
                //printf("deltaMin %i\n",i);
                //print_domains(n,x_domain,y_domain);
                stop=0;
            }
            if(getMax(n,y_domain,i)!=getMax(n,old_y_domain,i)){
                deltaMax(i,n,x_domain,y_domain,xpl,ypl,xPy,yPx,yub);
                //printf("deltaMax %i\n",i);
                //print_domains(n,x_domain,y_domain);
                stop=0;
            }
            for(int k=getMin(n,x_domain,i)+1;k<=getMax(n,old_x_domain,i);k++){
                if(getDomainBit(x_domain,i,k,n)!=getDomainBit(old_x_domain,i,k,n)){
                    removeValue(i,k,n,x_domain,y_domain,xpl,ypl,xPy,yPx);
                    //printf("removeValue %i %i\n",i,k);
                    //print_domains(n,x_domain,y_domain);
                    stop=0;
                }
            }
            //Applies remove value on the women too (this is missing from the original paper)
            for(int k=getMin(n,old_y_domain,i);k<getMax(n,y_domain,i);k++){
                if(getDomainBit(y_domain,i,k,n)!=getDomainBit(old_y_domain,i,k,n)){
                    removeValue(i,k,n,y_domain,x_domain,ypl,xpl,yPx,xPy);
                    stop=0;
                }
            }
        }

        if(stop){
            break;
        }

        //printf("I have not stopped!\n");

        //updates old domains
        for(int i=0;i<(n * n) / 32 + (n % 32 != 0);i++){
            old_x_domain[i]=prev_x_domain[i];
            old_y_domain[i]=prev_y_domain[i];
            prev_x_domain[i]=x_domain[i];
            prev_y_domain[i]=y_domain[i];
        }
    }
    
    //Frees memory and closes
    free(xPy);
    free(yPx);
    free(xlb);
    free(yub);
    free(old_x_domain);
    free(old_y_domain);
    free(prev_x_domain);
    free(prev_y_domain);
    
    return 0;
}

/*
    CUDA CONSTRAINT
*/

int cuda_constraint(int n, int *xpl, int *ypl, uint32_t *x_domain, uint32_t *y_domain) {
    int temp, *temp_p;

    //Builds the reverse matrixes
    int *xPy, *yPx;
    xPy = (int *)malloc(n * n * sizeof(int));
    yPx = (int *)malloc(n * n * sizeof(int));
    build_reverse_matrix(n,xpl,xPy);
    build_reverse_matrix(n,ypl,yPx);

    //print_reverse_matrixes(n,xPy,yPx);

    //prepares other data and copies it into device memory
    int *d_xpl, *d_ypl, *d_xPy, *d_yPx;
    uint32_t *d_x_domain, *d_y_domain;

    HANDLE_ERROR(hipMalloc((void**)&d_xpl, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_ypl, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_xPy, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_yPx, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t)));
    HANDLE_ERROR(hipMalloc((void**)&d_y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t)));

    HANDLE_ERROR(hipMemcpy(d_xpl, xpl, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_ypl, ypl, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_xPy, xPy, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_yPx, yPx, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x_domain, x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_y_domain, y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyHostToDevice));

    int *array_mod_men, *array_mod_women, *array_min_mod_men, *stack_mod_men, *stack_mod_women, *stack_mod_min_men, *new_stack_mod_min_men, *length_men_stack, *length_women_stack, *length_min_men_stack, *new_length_min_men_stack;
	HANDLE_ERROR(hipHostAlloc((void**)&array_mod_men, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&array_mod_women, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&array_min_mod_men, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&stack_mod_men, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&stack_mod_women, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&stack_mod_min_men, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&new_stack_mod_min_men, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&length_men_stack, sizeof (int), hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&length_women_stack, sizeof (int), hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&length_min_men_stack, sizeof (int), hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&new_length_min_men_stack, sizeof (int), hipHostMallocMapped));
    *length_men_stack = n;
    *length_women_stack = n;
    *length_min_men_stack = 0; //for f1 we pretend that it's empty, then we fill it before f2
    *new_length_min_men_stack = 0;
    for (int i=0;i<n;i++){
        array_mod_men[i]=1;
        array_mod_women[i]=1;
        array_min_mod_men[i]=1;
        stack_mod_men[i]=i;
        stack_mod_women[i]=i;
        //stack_mod_min_men[i]=i;
    }
    int *d_array_mod_men, *d_array_mod_women, *d_array_min_mod_men, *d_stack_mod_men, *d_stack_mod_women, *d_stack_mod_min_men, *d_new_stack_mod_min_men, *d_length_men_stack, *d_length_women_stack, *d_length_min_men_stack, *d_new_length_min_men_stack;
    HANDLE_ERROR(hipHostGetDevicePointer(&d_array_mod_men, array_mod_men, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&d_array_mod_women, array_mod_women, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&d_array_min_mod_men, array_min_mod_men, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&d_stack_mod_men, stack_mod_men, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&d_stack_mod_women, stack_mod_women, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&d_stack_mod_min_men, stack_mod_min_men, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&d_new_stack_mod_min_men, new_stack_mod_min_men, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&d_length_men_stack, length_men_stack, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&d_length_women_stack, length_women_stack, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&d_length_min_men_stack, length_min_men_stack, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&d_new_length_min_men_stack, new_length_min_men_stack, 0));

    int *old_min_men, *old_max_men, *old_min_women, *old_max_women;
    int *d_old_min_men, *d_old_max_men, *d_old_min_women, *d_old_max_women;
    old_min_men = (int*)malloc(sizeof (int) * n);
    old_max_men = (int*)malloc(sizeof (int) * n);
    old_min_women = (int*)malloc(sizeof (int) * n);
    old_max_women = (int*)malloc(sizeof (int) * n);
    for(int i=0;i<n;i++){
        old_min_men[i]=0;
        old_min_women[i]=0;
        old_max_men[i]=n-1;
        old_max_women[i]=n-1;
    }
    HANDLE_ERROR(hipMalloc((void**)&d_old_min_men, sizeof(int) * n));
    HANDLE_ERROR(hipMalloc((void**)&d_old_max_men, sizeof(int) * n));
    HANDLE_ERROR(hipMalloc((void**)&d_old_min_women, sizeof(int) * n));
    HANDLE_ERROR(hipMalloc((void**)&d_old_max_women, sizeof(int) * n));
    HANDLE_ERROR(hipMemcpy(d_old_min_men, old_min_men, sizeof(int) * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_old_max_men, old_max_men, sizeof(int) * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_old_min_women, old_min_women, sizeof(int) * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_old_max_women, old_max_women, sizeof(int) * n, hipMemcpyHostToDevice));
	
    //  computes the vectors of maxes and mins
    //  it may or may not be more efficient to compute after make_domains_coherent, depending on the implementations of the maxes and mins of the domains in the solver
    int *min_men, *max_men, *min_women, *max_women;
    int *d_min_men, *d_max_men, *d_min_women, *d_max_women;
    min_men = (int*)malloc(sizeof (int) * n);
    max_men = (int*)malloc(sizeof (int) * n);
    min_women = (int*)malloc(sizeof (int) * n);
    max_women = (int*)malloc(sizeof (int) * n);
    for(int i=0;i<n;i++){
        //initializes for the case of empty domains
        min_men[i]=n;
        min_women[i]=n;
        max_men[i]=n-1;
        max_women[i]=n-1;
        temp=0;
        while(temp<n&&getDomainBit(x_domain,i,temp,n)==0){
            temp++;
        }
        min_men[i]=temp;
        if(temp==n){//empty domain
            max_men[i]=n-1;
        }else{
            temp=n-1;
            while(getDomainBit(x_domain,i,temp,n)==0){//doesn't need to check for temp>=0 since we know it's not empty
                //printf("Found empty for man %i value %i",i,temp);
                temp--;
            }
            max_men[i]=temp;
            //printf("max men[%i]=%i\n",i,max_men[i]);
        }
        temp=0;
        while(temp<n&&getDomainBit(y_domain,i,temp,n)==0){
            temp++;
        }
        min_women[i]=temp;
        if(temp==n){//empty domain
            max_women[i]=n-1;
        }else{
            temp=n-1;
            while(getDomainBit(y_domain,i,temp,n)==0){//doesn't need to check for temp>=0 since we know it's not empty
                temp--;
            }
            max_women[i]=temp;
        }
    }
    HANDLE_ERROR(hipMalloc((void**)&d_min_men, sizeof(int) * n));
    HANDLE_ERROR(hipMalloc((void**)&d_max_men, sizeof(int) * n));
    HANDLE_ERROR(hipMalloc((void**)&d_min_women, sizeof(int) * n));
    HANDLE_ERROR(hipMalloc((void**)&d_max_women, sizeof(int) * n));
    HANDLE_ERROR(hipMemcpy(d_min_men, min_men, sizeof(int) * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_max_men, max_men, sizeof(int) * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_min_women, min_women, sizeof(int) * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_max_women, max_women, sizeof(int) * n, hipMemcpyHostToDevice));

    //runs kernels
    int device;
    hipGetDevice(&device);

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    int n_SMP = props.multiProcessorCount;
    int n_threads = *length_men_stack + *length_women_stack;
    int n_blocks, block_size;
    get_block_number_and_dimension(n_threads,n_SMP,&block_size,&n_blocks);
    
    //stampare i valori di sopra (debug)
    //printf("Prima di lancio di f1: %i, %i, %i\n", n_threads, n_blocks,block_size);
    
    make_domains_coherent<<<n_blocks,block_size>>>(n,d_xpl,d_ypl,d_xPy,d_yPx,d_x_domain,d_y_domain,d_array_mod_men, d_array_mod_women, d_array_min_mod_men, d_stack_mod_men, d_stack_mod_women, d_length_men_stack, d_length_women_stack, d_stack_mod_min_men, d_length_min_men_stack, d_old_min_men, d_old_max_men, d_old_min_women, d_old_max_women);
    hipDeviceSynchronize();

    //debug
    //printf("After f1:\n");
    //HANDLE_ERROR(hipMemcpy(x_domain, d_x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));
    //HANDLE_ERROR(hipMemcpy(y_domain, d_y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));
    //print_domains(n,x_domain,y_domain);
    //debug

    //empties array_min_mod_men
    HANDLE_ERROR(hipMemset(d_array_min_mod_men,0,sizeof(int)*n));

    //completely fills min_men_stack
    *length_min_men_stack = n;
    for (int i=0;i<n;i++){
        stack_mod_min_men[i]=i;
    }

    n_threads = *length_min_men_stack;
    get_block_number_and_dimension(n_threads,n_SMP,&block_size,&n_blocks);
    
    //DEBUG
    //n_blocks = n_threads;
    //block_size =1;
    //DEBUG
    
    //printf("Prima di lancio di f2: %i, %i, %i\n", n_threads, n_blocks,block_size);

    //printf("new_length_min_men_stack vale: %i\n",*new_length_min_men_stack);
    apply_sm_constraint<<<n_blocks,block_size>>>(n,d_xpl,d_ypl,d_xPy,d_yPx,d_x_domain,d_y_domain, d_array_min_mod_men, d_stack_mod_min_men, d_length_min_men_stack, d_new_stack_mod_min_men, d_new_length_min_men_stack, d_old_min_men, d_old_max_men, d_old_min_women, d_old_max_women, d_min_men, d_max_men, d_min_women, d_max_women);
    hipDeviceSynchronize();
    //printf("new_length_min_men_stack vale: %i\n",*new_length_min_men_stack);
    while(*new_length_min_men_stack!=0){
        //debug
        //printf("After f1:\n");
        //HANDLE_ERROR(hipMemcpy(x_domain, d_x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));
        //HANDLE_ERROR(hipMemcpy(y_domain, d_y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));
        //print_domains(n,x_domain,y_domain);
        //debug

        HANDLE_ERROR(hipMemset(d_array_min_mod_men,0,sizeof(int)*n));
        *length_min_men_stack = *new_length_min_men_stack;
        *new_length_min_men_stack = 0;
        temp_p = d_new_stack_mod_min_men;
        d_new_stack_mod_min_men = d_stack_mod_min_men;
        d_stack_mod_min_men = temp_p;
        //temp_p = new_stack_mod_min_men;
        //new_stack_mod_min_men = stack_mod_min_men;
        //stack_mod_min_men = temp_p;
        n_threads = *length_min_men_stack;
        get_block_number_and_dimension(n_threads,n_SMP,&block_size,&n_blocks);
        apply_sm_constraint<<<n_blocks,block_size>>>(n,d_xpl,d_ypl,d_xPy,d_yPx,d_x_domain,d_y_domain, d_array_min_mod_men, d_stack_mod_min_men, d_length_min_men_stack, d_new_stack_mod_min_men, d_new_length_min_men_stack, d_old_min_men, d_old_max_men, d_old_min_women, d_old_max_women, d_min_men, d_max_men, d_min_women, d_max_women);
        hipDeviceSynchronize();
    }

    //debug
    //printf("After f2:\n");
    //HANDLE_ERROR(hipMemcpy(x_domain, d_x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));
    //HANDLE_ERROR(hipMemcpy(y_domain, d_y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));
    //print_domains(n,x_domain,y_domain);
    //debug

    n_threads = n;
    get_block_number_and_dimension(n_threads,n_SMP,&block_size,&n_blocks);
    finalize_changes<<<n_blocks,block_size>>>(n,d_xpl,d_ypl,d_xPy,d_yPx,d_x_domain,d_y_domain,d_array_mod_men, d_array_mod_women, d_array_min_mod_men, d_stack_mod_men, d_stack_mod_women, d_length_men_stack, d_length_women_stack, d_stack_mod_min_men, d_length_min_men_stack, d_old_min_men, d_old_max_men, d_old_min_women, d_old_max_women, d_min_men, d_max_men, d_min_women, d_max_women);

    //copies from device memory
    HANDLE_ERROR(hipMemcpy(x_domain, d_x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(y_domain, d_y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(old_min_men, d_old_min_men, sizeof(int) * n, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(old_max_men, d_old_max_men, sizeof(int) * n, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(old_min_women, d_old_min_women, sizeof(int) * n, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(old_max_women, d_old_max_women, sizeof(int) * n, hipMemcpyDeviceToHost));

    //sets the lenghts to 0 (useless in this mockup)
    *length_men_stack = 0;
    *length_women_stack = 0;
    *length_min_men_stack = 0;

    //frees device memory
    HANDLE_ERROR(hipFree(d_xpl));
	HANDLE_ERROR(hipFree(d_ypl));
	HANDLE_ERROR(hipFree(d_xPy));
	HANDLE_ERROR(hipFree(d_yPx));
	HANDLE_ERROR(hipFree(d_x_domain));
	HANDLE_ERROR(hipFree(d_y_domain));
    
    HANDLE_ERROR(hipHostFree(array_mod_men));
    HANDLE_ERROR(hipHostFree(array_mod_women));
    HANDLE_ERROR(hipHostFree(array_min_mod_men));
    HANDLE_ERROR(hipHostFree(stack_mod_men));
    HANDLE_ERROR(hipHostFree(stack_mod_women));
    HANDLE_ERROR(hipHostFree(stack_mod_min_men));
    HANDLE_ERROR(hipHostFree(new_stack_mod_min_men));
    HANDLE_ERROR(hipHostFree(length_men_stack));
    HANDLE_ERROR(hipHostFree(length_women_stack));
    HANDLE_ERROR(hipHostFree(length_min_men_stack));
    HANDLE_ERROR(hipHostFree(new_length_min_men_stack));

    HANDLE_ERROR(hipFree(d_old_min_men));
    HANDLE_ERROR(hipFree(d_old_max_men));
    HANDLE_ERROR(hipFree(d_old_min_women));
    HANDLE_ERROR(hipFree(d_old_max_women));
    HANDLE_ERROR(hipFree(d_min_men));
    HANDLE_ERROR(hipFree(d_max_men));
    HANDLE_ERROR(hipFree(d_min_women));
    HANDLE_ERROR(hipFree(d_max_women));

    
    for(int i=0;i<n;i++){
        if(getMin(n,x_domain,i)<=n&&getMin(n,x_domain,i)!=old_min_men[i]){
            printf("Mistake in man %i! Min is %i and old_min is %i!\n",i,getMin(n,x_domain,i),old_min_men[i]);
        }
    }
    //debug
    /*for(int i=0;i<n;i++){
        if(getMax(n,x_domain,i)>0&&getMax(n,x_domain,i)!=old_max_men[i]){
            printf("Mistake in man %i! Max is %i and old_max is %i!\n",i,getMax(n,x_domain,i),old_max_men[i]);
        }
    }

    
    for(int i=0;i<n;i++){
        if(getMin(n,y_domain,i)<=n&&getMin(n,y_domain,i)!=old_min_women[i]){
            printf("Mistake in woman %i! Min is %i and old_min is %i!\n",i,getMin(n,y_domain,i),old_min_women[i]);
        }
    }
    for(int i=0;i<n;i++){
        if(getMax(n,y_domain,i)>0&&getMax(n,y_domain,i)!=old_max_women[i]){
            printf("Mistake in woman %i! Max is %i and old_max is %i!\n",i,getMax(n,y_domain,i),old_max_women[i]);
        }
    }*/
    //debug
    
    
    //Frees memory and closes
    free(xPy);
    free(yPx);
    free(old_min_men);
    free(old_min_women);
    free(old_max_men);
    free(old_max_women);
    free(min_men);
    free(max_men);
    free(min_women);
    free(max_women);

    //checks if there's an empty domain
    int emptyX, emptyY;
    for(int i=0;i<n;i++){
        emptyX = true;
        emptyY = true;
        for(int j=0;j<n;j++){
            if(getDomainBit(x_domain,i,j,n)){
                emptyX=false;
            }
            if(getDomainBit(y_domain,i,j,n)){
                emptyY=false;
            }
        }
        if(emptyX || emptyY){
            //printf("Empty.\n");
            return -1;
        }
    }
    return 0;
}

/*
    Computes the appropriate block size and number of blocks based on the number of threads required and the number of SMPs
*/
void get_block_number_and_dimension(int n_threads, int n_SMP, int *block_size, int *n_blocks){
    if (n_threads/n_SMP >= 32){ //at least one warp per SMP
        *n_blocks = n_SMP;
        *block_size = (n_threads + *n_blocks - 1) / *n_blocks;
    } else { //less than one warp per SMP
        *block_size = 32;
        *n_blocks = (n_threads + 31) / 32;
    }
}

/*
    SHARED UTILS
*/

void build_reverse_matrix(int n,int *zpl, int *zPz){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            zPz[i*n+zpl[i*n+j]]=j;
        }
    }
}

/*
    Creates a random n*n preference matrix
    Uses global rng defined in the pcg-basic-c library
*/
int* make_random_preference_matrix(int n){
	int* preference_matrix = (int*)malloc(sizeof (int) * n * n);
	int* numbers = (int*)malloc(sizeof (int) * n);
	int x,t;
	for(int j=0;j<n;j++){
		numbers[j]=j;
	}
	for(int i=0;i<n;i++){
		for(int j=0;j<n-1;j++){
			x = (int) pcg32_boundedrand(n-j);
			preference_matrix[i*n+j]=numbers[x];
            t = numbers[x];
			numbers[x]=numbers[n-j-1];
            //moves the numbers instead of overwriting them, so to not have to initialize the vector everytime
			numbers[n-j-1]=t;
		}
		preference_matrix[i*n+n-1]=numbers[0];
	}
	free(numbers);
	return preference_matrix;
}

/*
    Creates a full domain of given size
    Takes a pointer to a properly allocated domain
*/
void make_full_domain(int n, uint32_t *domain){
    for(int i=0;i<(n * n) / 32 + (n % 32 != 0);i++){
        domain[i] = 4294967295;
    }
}

/*
    Creates a randomized domain of given size
    Uses global rng defined in the pcg-basic-c library
    Takes a pointer to a properly allocated domain
*/
void make_partial_domain(int n, uint32_t *domain){
    for(int i=0;i<(n * n) / 32 + (n % 32 != 0);i++){
        domain[i] = pcg32_random() | pcg32_random() | pcg32_random() | pcg32_random() | pcg32_random() | pcg32_random() | pcg32_random() | pcg32_random() | pcg32_random() | pcg32_random();
    }
}

/*
    Taken a domain, it creates a copy
*/
void clone_domain(int n, uint32_t *old_domain, uint32_t *new_domain){
    for(int i=0;i<(n * n) / 32 + (n % 32 != 0);i++){
        new_domain[i] = old_domain[i];
    }
}

/*
    Compares two domains
    For performance reasons, it includes in the comparison the ending part of the allocated memory that doesn't contain any info.
    The correctness of this function depends on how the domains were created.
*/
int compare_domains(int n, uint32_t *d1, uint32_t *d2){
    for(int i=0;i<(n * n) / 32 + (n % 32 != 0);i++){
        if(d1[i]!=d2[i]){
            return 0;
        }
    }
    return 1;
}