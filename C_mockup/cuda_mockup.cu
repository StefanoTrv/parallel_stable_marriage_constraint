#include <stdio.h>
#include "utils/io.c"
#include "utils/printer.c"
#include "domain_functions.c"
#include "utils\error_handler.cu"
#include "cuda_costraint.cu"

void build_reverse_matrix(int,int*, int*);

int main(int argc, char *argv[]) {
    // Get file path from command line arguments or uses a default value
    char *file_path;
    if (argc > 1) {
        file_path = argv[1];
    } else {
        file_path = "input.txt";
    }

    //Reads input data
    int n;
    int *xpl, *ypl;
    uint32_t *x_domain, *y_domain;

    parse_input(file_path, &n, &xpl, &ypl, &x_domain, &y_domain);

    //If the input didn't include the domains, all the domains are initialized as full
    if(x_domain==NULL){
        x_domain = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
        y_domain = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
        // Read x_domain
        for(int i=0;i<(n * n) / 32 + (n % 32 != 0);i++){
            x_domain[i] = 4294967295;
            y_domain[i] = 4294967295;
        }
    }

    print_preference_lists(n,xpl,ypl);

    print_domains(n, x_domain, y_domain);

    //Builds the reverse matrixes
    int *xPy, *yPx;
    xPy = (int *)malloc(n * n * sizeof(int));
    yPx = (int *)malloc(n * n * sizeof(int));
    build_reverse_matrix(n,xpl,xPy);
    build_reverse_matrix(n,ypl,yPx);

    print_reverse_matrixes(n,xPy,yPx);

    //prepares other data and copies into device memory
    int *d_xpl, *d_ypl, *d_xPy, *d_yPx;
    uint32_t *d_x_domain, *d_y_domain;

    HANDLE_ERROR(hipMalloc((void**)&d_xpl, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_ypl, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_xPy, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_yPx, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t)));
    HANDLE_ERROR(hipMalloc((void**)&d_y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t)));

    HANDLE_ERROR(hipMemcpy(d_xpl, xpl, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_ypl, ypl, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_xPy, xPy, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_yPx, yPx, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x_domain, x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_y_domain, y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyHostToDevice));

    int *array_mod_men, *array_mod_women, *array_min_mod_men, *stack_mod_men, *stack_mod_women, *stack_mod_min_men;
    int length_men_stack, length_women_stack, length_min_men_stack;
    length_men_stack = n;
    length_women_stack = n;
    length_min_men_stack = n;
	HANDLE_ERROR(hipHostAlloc((void**)&array_mod_men, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&array_mod_women, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&array_min_mod_men, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&stack_mod_men, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&stack_mod_women, sizeof (int) * n, hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&stack_mod_min_men, sizeof (int) * n, hipHostMallocMapped));
    for (int i=0;i<n;i++){
        array_mod_men[i]=1;
        array_mod_women[i]=1;
        array_min_mod_men[i]=1;
        stack_mod_men[i]=i;
        stack_mod_women[i]=i;
        stack_mod_min_men[i]=i;
    }

    int *old_min_man, *old_max_man, *old_min_woman, *old_max_woman;
    int *d_old_min_man, *d_old_max_man, *d_old_min_woman, *d_old_max_woman;
    old_min_man = (int*)malloc(sizeof (int) * n);
    old_max_man = (int*)malloc(sizeof (int) * n);
    old_min_woman = (int*)malloc(sizeof (int) * n);
    old_max_woman = (int*)malloc(sizeof (int) * n);
    for(int i=0;i<n;i++){
        old_min_man[i]=0;
        old_min_woman[i]=0;
        old_max_man[i]=n-1;
        old_max_woman[i]=n-1;
    }
    HANDLE_ERROR(hipMalloc((void**)&d_old_min_man, sizeof(int) * n));
    HANDLE_ERROR(hipMalloc((void**)&d_old_max_man, sizeof(int) * n));
    HANDLE_ERROR(hipMalloc((void**)&d_old_min_woman, sizeof(int) * n));
    HANDLE_ERROR(hipMalloc((void**)&d_old_max_woman, sizeof(int) * n));
    HANDLE_ERROR(hipMemcpy(d_old_min_man, old_min_man, sizeof(int) * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_old_max_man, old_max_man, sizeof(int) * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_old_min_woman, old_min_woman, sizeof(int) * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_old_max_woman, old_max_woman, sizeof(int) * n, hipMemcpyHostToDevice));
	

    //runs kernels
    int device;
    hipGetDevice(&device);

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    int n_blocks = props.multiProcessorCount;
    int block_size = (n + n_blocks - 1) / n_blocks;
    
    my_kernel<<<n_blocks,block_size>>>(n,d_xpl,d_ypl,d_xPy,d_yPx,d_x_domain,d_y_domain);

    //copies from device memory
    HANDLE_ERROR(hipMemcpy(x_domain, d_x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(y_domain, d_y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));

    //frees device memory
    HANDLE_ERROR(hipFree(d_xpl));
	HANDLE_ERROR(hipFree(d_ypl));
	HANDLE_ERROR(hipFree(d_xPy));
	HANDLE_ERROR(hipFree(d_yPx));
	HANDLE_ERROR(hipFree(d_x_domain));
	HANDLE_ERROR(hipFree(d_y_domain));
    
    HANDLE_ERROR(hipHostFree(array_mod_men));
    HANDLE_ERROR(hipHostFree(array_mod_women));
    HANDLE_ERROR(hipHostFree(array_min_mod_men));
    HANDLE_ERROR(hipHostFree(stack_mod_men));
    HANDLE_ERROR(hipHostFree(stack_mod_women));
    HANDLE_ERROR(hipHostFree(stack_mod_min_men));

    HANDLE_ERROR(hipFree(d_old_min_man));
    HANDLE_ERROR(hipFree(d_old_max_man));
    HANDLE_ERROR(hipFree(d_old_min_woman));
    HANDLE_ERROR(hipFree(d_old_max_woman));
	

    print_domains(n,x_domain,y_domain);

    printf("Men best:\n");
    for(int i = 0;i<n;i++){
        printf("%i",xpl[i*n+getMin(n,x_domain,i)]);
    }
    
    
    //Frees memory and closes
    free(xpl);
    free(ypl);
    free(x_domain);
    free(y_domain);
    free(xPy);
    free(yPx);
    free(old_min_man);
    free(old_min_woman);
    free(old_max_man);
    free(old_max_woman);

    printf("\nClosing...");

    return 0;
}

void build_reverse_matrix(int n,int *zpl, int *zPz){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            zPz[i*n+zpl[i*n+j]]=j;
        }
    }
}