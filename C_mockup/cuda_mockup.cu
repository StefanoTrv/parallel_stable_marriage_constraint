#include <stdio.h>
#include "utils/io.c"
#include "utils/printer.c"
#include "domain_functions.c"
#include "utils\error_handler.cu"
#include "cuda_costraint.cu"

void build_reverse_matrix(int,int*, int*);

int main(int argc, char *argv[]) {
    // Get file path from command line arguments or uses a default value
    char *file_path;
    if (argc > 1) {
        file_path = argv[1];
    } else {
        file_path = "input.txt";
    }

    //Reads input data
    int n;
    int *xpl, *ypl;
    uint32_t *x_domain, *y_domain;

    parse_input(file_path, &n, &xpl, &ypl, &x_domain, &y_domain);

    //If the input didn't include the domains, all the domains are initialized as full
    if(x_domain==NULL){
        x_domain = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
        y_domain = (uint32_t *)malloc(((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t));
        // Read x_domain
        for(int i=0;i<(n * n) / 32 + (n % 32 != 0);i++){
            x_domain[i] = 4294967295;
            y_domain[i] = 4294967295;
        }
    }

    print_preference_lists(n,xpl,ypl);

    print_domains(n, x_domain, y_domain);

    //Builds the reverse matrixes
    int *xPy, *yPx;
    xPy = (int *)malloc(n * n * sizeof(int));
    yPx = (int *)malloc(n * n * sizeof(int));
    build_reverse_matrix(n,xpl,xPy);
    build_reverse_matrix(n,ypl,yPx);

    print_reverse_matrixes(n,xPy,yPx);

    //copies into device memory
    int *d_xpl, *d_ypl, *d_xPy, *d_yPx;
    uint32_t *d_x_domain, *d_y_domain;

    HANDLE_ERROR(hipMalloc((void**)&d_xpl, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_ypl, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_xPy, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_yPx, sizeof(int) * n * n));
    HANDLE_ERROR(hipMalloc((void**)&d_x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t)));
    HANDLE_ERROR(hipMalloc((void**)&d_y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t)));

    HANDLE_ERROR(hipMemcpy(d_xpl, xpl, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_ypl, ypl, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_xPy, xPy, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_yPx, yPx, sizeof(int) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x_domain, x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_y_domain, y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyHostToDevice));

    //runs kernels
    int device;
    hipGetDevice(&device);

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    int n_blocks = props.multiProcessorCount;
    int block_size = (n + n_blocks - 1) / n_blocks;
    
    my_kernel<<<n_blocks,block_size>>>();

    //copies from device memory
    HANDLE_ERROR(hipMemcpy(x_domain, d_x_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(y_domain, d_y_domain, ((n * n) / 32 + (n % 32 != 0)) * sizeof(uint32_t), hipMemcpyDeviceToHost));

    //frees device memory
    HANDLE_ERROR(hipFree(d_xpl));
	HANDLE_ERROR(hipFree(d_ypl));
	HANDLE_ERROR(hipFree(d_xPy));
	HANDLE_ERROR(hipFree(d_yPx));
	HANDLE_ERROR(hipFree(d_x_domain));
	HANDLE_ERROR(hipFree(d_y_domain));
	

    print_domains(n,x_domain,y_domain);

    printf("Men best:\n");
    for(int i = 0;i<n;i++){
        printf("%i",xpl[i*n+getMin(n,x_domain,i)]);
    }
    
    
    //Frees memory and closes
    free(xpl);
    free(ypl);
    free(x_domain);
    free(y_domain);
    free(xPy);
    free(yPx);

    return 0;
}

void build_reverse_matrix(int n,int *zpl, int *zPz){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            zPz[i*n+zpl[i*n+j]]=j;
        }
    }
}