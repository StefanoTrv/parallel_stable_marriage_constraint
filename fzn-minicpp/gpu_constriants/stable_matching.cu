#include "hip/hip_runtime.h"
#include "stable_matching.cuh"
#include "error_handler.cuh"

const uint32_t UNS_ONE = 1;

__global__ void make_domains_coherent(int n, int* xpl, int* ypl, int* xPy, int* yPx, uint32_t* x_domain, uint32_t* y_domain, int* stack_mod_men, int* stack_mod_women, int length_men_stack, int length_women_stack, int* stack_mod_min_men, int* length_min_men_stack, int* old_min_men, int* old_max_men, int* old_min_women, int* old_max_women);
__global__ void apply_sm_constraint(int n, int* xpl, int* ypl, int* xPy, int* yPx, uint32_t* x_domain, uint32_t* y_domain, int* array_min_mod_men, int* stack_mod_min_men, int* length_min_men_stack, int* new_stack_mod_min_men, int* new_length_min_men_stack, int* old_min_men, int* max_men, int* max_women);
__global__ void finalize_changes(int n, uint32_t* x_domain, uint32_t* y_domain, int* old_min_men, int* old_max_men, int* old_min_women, int* old_max_women, int* max_men, int* min_women, int* max_women);

/*void print_domains(int n, uint32_t* x_dom, uint32_t* y_dom) {
    int index, offset;
    // Men
    printf("\nMen domains:\n");
    printf("_\t");
    for (int i = 0; i < n; i++) {
        printf("%d\t", i);
    }
    for (int i = 0; i < n; i++) {
        printf("\n%d:\t", i);
        for (int j = 0; j < n; j++) {
            index = i*n+j;
            offset = index % 32;
            printf("%d\t", (x_dom[index/32] << offset) >> (sizeof (int)*8 - 1));
        }
        printf("\n");
    }
    
    printf("\n");

    // Women
    printf("\nWomen domains:\n");
    printf("_\t");
    for (int i = 0; i < n; i++) {
        printf("%d\t", i);
    }
    for (int i = 0; i < n; i++) {
        printf("\n%d:\t", i);
        for (int j = 0; j < n; j++) {
            index = i*n+j;
            offset = index % 32;
            printf("%d\t", (y_dom[index/32] << offset) >> (sizeof (int)*8 - 1));
        }
        printf("\n");
    }
            
    printf("\n\n");
}*/

StableMatchingGPU::StableMatchingGPU(std::vector<var<int>::Ptr> & m, std::vector<var<int>::Ptr> & w, std::vector<std::vector<int>> const & mpl, std::vector<std::vector<int>> const & wpl) :
    Constraint(m[0]->getSolver()), _x(m), _y(w), _xpl_vector(mpl), _ypl_vector(wpl)
{
    setPriority(CLOW);
    hipStreamCreate(&_stream);

    // Get the size of the problem instance
    _n = static_cast<int>(_x.size());

    //Domain device memory allocation
    HANDLE_ERROR(hipMalloc((void**)&_d_x_domain, ((_n * _n) / 32 + (_n % 32 != 0)) * 2 * sizeof(uint32_t)));
    _d_y_domain = _d_x_domain + (_n * _n) / 32 + (_n % 32 != 0);

    //Host memory allocation
    _x_domain = (uint32_t *)malloc(((_n * _n) / 32 + (_n % 32 != 0)) * 2 * sizeof(uint32_t));
    _y_domain = _x_domain + ((_n * _n) / 32 + (_n % 32 != 0));
    _xpl = (int *)malloc((_n * _n * 4 + _n * 10 + 2) * sizeof(int));
    _ypl = _xpl + (_n* _n);
    _xPy = _ypl + (_n* _n);
    _yPx = _xPy + (_n* _n);
    _stack_mod_men = _yPx + (_n* _n);
    _stack_mod_women = _stack_mod_men + _n;
    _stack_mod_min_men = _stack_mod_women + _n;
    _old_min_men = _stack_mod_min_men + _n;
    _old_max_men = _old_min_men + _n;
    _old_min_women = _old_max_men + _n;
    _old_max_women = _old_min_women + _n;
    _max_men = _old_max_women + _n;
    _min_women = _max_men + _n;
    _max_women = _min_women + _n;
    _length_min_men_stack = _max_women + _n;
    _new_length_min_men_stack = _length_min_men_stack + 1;

    //Device memory allocation
    HANDLE_ERROR(hipMalloc((void**)&_d_xpl, sizeof(int) * (_n * _n * 4 + _n * 12 + 2)));
    _d_ypl = _d_xpl + _n * _n;
    _d_xPy = _d_ypl + _n * _n;
    _d_yPx = _d_xPy + _n * _n;
    _d_stack_mod_men = _d_yPx + _n * _n;
    _d_stack_mod_women = _d_stack_mod_men + _n;
    _d_stack_mod_min_men = _d_stack_mod_women + _n;
    _d_old_min_men = _d_stack_mod_min_men + _n;
    _d_old_max_men = _d_old_min_men + _n;
    _d_old_min_women = _d_old_max_men + _n;
    _d_old_max_women = _d_old_min_women + _n;
    _d_max_men = _d_old_max_women + _n;
    _d_min_women = _d_max_men + _n;
    _d_max_women = _d_min_women + _n;
    _d_length_min_men_stack = _d_max_women + _n;
    _d_new_length_min_men_stack = _d_length_min_men_stack + 1;
    _d_new_stack_mod_min_men = _d_new_length_min_men_stack + 1;
    _d_array_min_mod_men = _d_new_stack_mod_min_men + _n;
    
    //Initialize trailable vectors
    for (int i = 0; i < _n; i++){
        _old_max_women_trail.push_back(trail<int>(m[0]->getSolver()->getStateManager(), _n-1));
        _old_min_women_trail.push_back(trail<int>(m[0]->getSolver()->getStateManager(), 0));
        _old_max_men_trail.push_back(trail<int>(m[0]->getSolver()->getStateManager(), _n-1));
        _old_min_men_trail.push_back(trail<int>(m[0]->getSolver()->getStateManager(), 0));
    }

    //Prepares the data structures that won't be modified by post()
    copyPreferenceMatrix(_xpl_vector,_xpl);
    copyPreferenceMatrix(_ypl_vector,_ypl);
    buildReverseMatrix(_xpl_vector,_xPy);
    buildReverseMatrix(_ypl_vector,_yPx);
    *_length_min_men_stack = 0; //for f1 we pretend that it's empty, then we fill it before f2
    *_new_length_min_men_stack = 0;
    for (int i=0;i<_n;i++){
        _stack_mod_men[i]=i;
        _stack_mod_women[i]=i;
    }
    for(int i=0;i<_n;i++){
        _old_min_men[i]=0;
        _old_min_women[i]=0;
        _old_max_men[i]=_n-1;
        _old_max_women[i]=_n-1;
    }

    //Copy the data structures that won't be modified by post()
    HANDLE_ERROR(hipMemcpyAsync(_d_xpl, _xpl, (_n * _n * 4 + _n * 7) * sizeof(int), hipMemcpyHostToDevice, _stream));

    //Get number of SMPs in the device
    int device;
    hipGetDevice(&device);
    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    _n_SMP = props.multiProcessorCount;
}

void StableMatchingGPU::post(){
    int _length_men_stack, _length_women_stack;
    _length_men_stack = _n;
    _length_women_stack = _n;

    //Finds current maxes and mins
    for(int i=0; i<_n; i++){
        _min_women[i]=_y[i]->min();
        _max_men[i]=_x[i]->max();
        _max_women[i]=_y[i]->max();
    }

    //Copies the remaining data structures
    HANDLE_ERROR(hipMemcpyAsync(_d_max_men, _max_men, (_n * 3 + 2) * sizeof(int), hipMemcpyHostToDevice, _stream));

    //Initializes the update_counters, which allow for baktracking identification
    _propagation_counter_trail = trail<int>(_x[0]->getSolver()->getStateManager(), 0);
    _propagation_counter = 1;

    //Initializes _x_old_sizes and _y_old_sizes with wrong sizes, to force dumping of all variables (redundant, because _propagation_counter!=_propagation_counter_trail)
    for (int i=0; i<_n; i++)
    {
        _x_old_sizes.push_back(trail<int>(_x[0]->getSolver()->getStateManager(), 0));
        _y_old_sizes.push_back(trail<int>(_x[0]->getSolver()->getStateManager(), 0));
    }

    //Copy the domains
    dumpDomainsToBitset(_x, _x_domain, _old_min_men, _old_max_men,_x_old_sizes);
    dumpDomainsToBitset(_y, _y_domain, _old_min_women, _old_max_women,_y_old_sizes);
    HANDLE_ERROR(hipMemcpyAsync(_d_x_domain, _x_domain, ((_n * _n) / 32 + (_n % 32 != 0)) * 2 * sizeof(uint32_t), hipMemcpyHostToDevice, _stream));

    /*
        Excute kernels
    */

    // Fun1
    int n_threads = _length_men_stack + _length_women_stack;
    int n_blocks, block_size;
    getBlockNumberAndDimension(n_threads,&block_size,&n_blocks);
    make_domains_coherent<<<n_blocks,block_size,0,_stream>>>(_n,_d_xpl,_d_ypl,_d_xPy,_d_yPx,_d_x_domain,_d_y_domain, _d_stack_mod_men, _d_stack_mod_women, _length_men_stack, _length_women_stack, _d_stack_mod_min_men, _d_length_min_men_stack, _d_old_min_men, _d_old_max_men, _d_old_min_women, _d_old_max_women);

    //Fun2
    //  empties d_array_min_mod_men
    HANDLE_ERROR(hipMemsetAsync(_d_array_min_mod_men,0,sizeof(int)*_n, _stream));

    //  completely fills min_men_stack
    *_length_min_men_stack = _n;
    for (int i=0;i<_n;i++){
        _stack_mod_min_men[i]=i;
    }
    HANDLE_ERROR(hipMemcpyAsync(_d_stack_mod_min_men, _stack_mod_min_men, sizeof(int) * _n, hipMemcpyHostToDevice, _stream));
    HANDLE_ERROR(hipMemcpyAsync(_d_length_min_men_stack, _length_min_men_stack, sizeof(int), hipMemcpyHostToDevice, _stream));

    iterateFun2();

    //Fun3
    n_threads = _n;
    getBlockNumberAndDimension(n_threads,&block_size,&n_blocks);
    finalize_changes<<<n_blocks,block_size,0,_stream>>>(_n,_d_x_domain,_d_y_domain, _d_old_min_men, _d_old_max_men, _d_old_min_women, _d_old_max_women, _d_max_men, _d_min_women, _d_max_women);

    /*
        Completed kernel execution (not yet synchronized)
    */

    //Update data structures and variables
    HANDLE_ERROR(hipMemcpyAsync(_x_domain, _d_x_domain, ((_n * _n) / 32 + (_n % 32 != 0)) * 2 * sizeof(uint32_t), hipMemcpyDeviceToHost, _stream));
    HANDLE_ERROR(hipMemcpyAsync(_old_min_men, _d_old_min_men, sizeof(int) * _n * 4, hipMemcpyDeviceToHost, _stream));
    hipStreamSynchronize(_stream);
    updateHostData();

    //Set propagation condition for variables
    //After first propagation to avoid useless propagate() call
    for (auto const & v : _x){
        v->propagateOnDomainChange(this);
    }
    for (auto const & v : _y){
        v->propagateOnDomainChange(this);
    }

    //Sets the counters
    _propagation_counter_trail = 1;
    _propagation_counter = 1;

    //Initializes _x_old_sizes and _y_old_sizes for the first propagation
    for (int i=0; i<_n; i++)
    {
        _x_old_sizes[i] = _x[i]->size();
        _y_old_sizes[i] = _y[i]->size();
    }
}

void StableMatchingGPU::propagate(){
    //Prepare other data structures
    int _length_men_stack, _length_women_stack;
    _length_men_stack = 0;
    _length_women_stack = 0;
    *_length_min_men_stack = 0;
    *_new_length_min_men_stack = 0;
    for(int i=0; i<_n; i++){
        if(_x[i]->size()!=_x_old_sizes[i]){ //if variable was modified (compares the sizes to avoid false positives given by changed())
            _stack_mod_men[_length_men_stack]=i;
            _length_men_stack++;
            if(_x[i]->min()!=_old_min_men_trail[i]){ //if min is changed (this comparison avoids false positives given by changedMin())
                _stack_mod_min_men[*_length_min_men_stack]=i;
                (*_length_min_men_stack)++;
            }
        }
        if(_y[i]->size()!=_y_old_sizes[i]){ //if variable was modified (compares the sizes to avoid false positives given by changed())
            _stack_mod_women[_length_women_stack]=i;
            _length_women_stack++;
        }
    }
    if(_length_men_stack+_length_women_stack==0){ //no variable needs to be updated: quits immediately
        return;
    }
    for(int i=0;i<_n;i++){
        _old_min_men[i]=_old_min_men_trail[i];
        _old_min_women[i]=_old_min_women_trail[i];
        _old_max_men[i]=_old_max_men_trail[i];
        _old_max_women[i]=_old_max_women_trail[i];
    }
    for(int i=0; i<_n; i++){
        _min_women[i]=_y[i]->min();
        _max_men[i]=_x[i]->max();
        _max_women[i]=_y[i]->max();
    }

    HANDLE_ERROR(hipMemcpyAsync(_d_stack_mod_men, _stack_mod_men, (_n * 10 + 2) * sizeof(int), hipMemcpyHostToDevice, _stream));

    //Copy domains to device
    dumpDomainsToBitset(_x, _x_domain, _old_min_men, _old_max_men, _x_old_sizes);
    dumpDomainsToBitset(_y, _y_domain, _old_min_women, _old_max_women, _y_old_sizes);
    HANDLE_ERROR(hipMemcpyAsync(_d_x_domain, _x_domain, ((_n * _n) / 32 + (_n % 32 != 0)) * 2 * sizeof(uint32_t), hipMemcpyHostToDevice, _stream));

    //Updates and increases counters
    _propagation_counter_trail += 1;
    _propagation_counter = _propagation_counter_trail;

    /*
        Excute kernels
    */

    // Fun1
    int n_threads = _length_men_stack + _length_women_stack;
    int n_blocks, block_size;
    getBlockNumberAndDimension(n_threads,&block_size,&n_blocks);
    make_domains_coherent<<<n_blocks,block_size,0,_stream>>>(_n,_d_xpl,_d_ypl,_d_xPy,_d_yPx,_d_x_domain,_d_y_domain, _d_stack_mod_men, _d_stack_mod_women, _length_men_stack, _length_women_stack, _d_stack_mod_min_men, _d_length_min_men_stack, _d_old_min_men, _d_old_max_men, _d_old_min_women, _d_old_max_women);

    //Fun2
    //  empties d_array_min_mod_men
    HANDLE_ERROR(hipMemsetAsync(_d_array_min_mod_men,0,sizeof(int)*_n, _stream));

    // updates _length_min_men_stack
    HANDLE_ERROR(hipMemcpyAsync(_length_min_men_stack, _d_length_min_men_stack, sizeof(int), hipMemcpyDeviceToHost, _stream));
    hipStreamSynchronize(_stream); // to be able to read _length_min_men_stack

    iterateFun2();

    //Fun3
    n_threads = _n;
    getBlockNumberAndDimension(n_threads,&block_size,&n_blocks);
    finalize_changes<<<n_blocks,block_size,0,_stream>>>(_n,_d_x_domain,_d_y_domain, _d_old_min_men, _d_old_max_men, _d_old_min_women, _d_old_max_women, _d_max_men, _d_min_women, _d_max_women);

    /*
        Completed kernel execution (not yet synchronized)
    */

    //Update data structures and variables
    HANDLE_ERROR(hipMemcpyAsync(_x_domain, _d_x_domain, ((_n * _n) / 32 + (_n % 32 != 0)) * 2 * sizeof(uint32_t), hipMemcpyDeviceToHost, _stream));
    HANDLE_ERROR(hipMemcpyAsync(_old_min_men, _d_old_min_men, sizeof(int) * _n * 4, hipMemcpyDeviceToHost, _stream));
    hipStreamSynchronize(_stream);
    updateHostData();

    //Updates old sizes
    for(int i=0; i<_n; i++){
        _x_old_sizes[i]=_x[i]->size();
        _y_old_sizes[i]=_y[i]->size();
    }
}

void StableMatchingGPU::buildReverseMatrix(std::vector<std::vector<int>> zpl, int *zPz){
    for(int i=0;i<_n;i++){
        for(int j=0;j<_n;j++){
            zPz[i*_n+zpl[i][j]]=j;
        }
    }
}

void StableMatchingGPU::copyPreferenceMatrix(std::vector<std::vector<int>> zpl_vec, int *zpl){
    for(int i=0;i<_n;i++){
        for(int j=0;j<_n;j++){
            zpl[i*_n+j]=zpl_vec[i][j];
        }
    }
}

void StableMatchingGPU::dumpDomainsToBitset(std::vector<var<int>::Ptr> vars, uint32_t* dom, int* old_mins, int* old_maxes, std::vector<trail<int>> old_sizes){
    int starting_bit,ending_bit;
    int starting_word, ending_word;
    int var_min, var_max;
    uint32_t mask;
    bool has_backtracked = _propagation_counter_trail != _propagation_counter;
    for(int i=0; i<_n; i++){
        //_has_backtracked allows the bitset to be correct even after backtracking
        if(vars[i]->size()==old_sizes[i] && !has_backtracked){ //checks size to see if variable has changed
            continue;
        }

        //Note: During the first dump, all the domains will be reset.
        var_min = old_mins[i];
        var_max = old_maxes[i];

        starting_bit = _n * i + var_min;
        starting_word = starting_bit/32;
        ending_bit = _n * i + var_max;
        ending_word = ending_bit/32; 

        if (starting_word == ending_word) {
            // Clear bits within a single word
            mask = 0xFFFFFFFF >> ((31 - ending_bit) % 32) << ((31 - ending_bit) + starting_bit) % 32 >> (starting_bit % 32);
            dom[starting_word] &= ~mask;
        } else {
            // Clear bits in the first word
            mask = 0xFFFFFFFF << (starting_bit % 32) >> (starting_bit % 32);
            dom[starting_word] &= ~mask;
        
            // Clear full words in between
            for (int w = starting_word + 1; w < ending_word; w++) {
                dom[w] = 0;
            }
        
            // Clear bits in the last word
            mask = 0xFFFFFFFF >> ((31 - ending_bit) % 32) << ((31 - ending_bit) % 32);
            dom[ending_word] &= ~mask;
        }
        starting_bit = _n * i + vars[i]->min();
        vars[i]->dumpWithOffset(vars[i]->min(),vars[i]->max(),dom + (starting_bit / 32),starting_bit % 32);
    }
}

int StableMatchingGPU::getBitHost(uint32_t* bitmap, int index){
    int offset = index % 32;
    return (bitmap[index/32] << offset) >> (sizeof (uint32_t)*8 - 1);
}

int StableMatchingGPU::getDomainBitHost(uint32_t* bitmap, int row, int column){
    return getBitHost(bitmap,row*_n+column);
}

// Updates the _olds trailable vectors and the variables with the data from the device
// Supposes that the copy from the device has already been completed
void StableMatchingGPU::updateHostData(){
    for (int i = 0; i < _n; i++){ //_olds
        _old_max_women_trail[i]=_old_max_women[i];
        _old_min_women_trail[i]=_old_min_women[i];
        _old_max_men_trail[i]=_old_max_men[i];
        _old_min_men_trail[i]=_old_min_men[i];
    }

    for (int i=0; i<_n; i++){ //_x and _y
        _x[i]->removeBelow(_old_min_men[i]);
        _x[i]->removeAbove(_old_max_men[i]);
        for(int j=_old_min_men[i]+1; j<_old_max_men[i]; j++){
            if(!getDomainBitHost(_x_domain,i,j)){
                _x[i]->remove(j);
            }
        }

        _y[i]->removeBelow(_old_min_women[i]);
        _y[i]->removeAbove(_old_max_women[i]);
        for(int j=_old_min_women[i]+1; j<_old_max_women[i]; j++){
            if(!getDomainBitHost(_y_domain,i,j)){
                _y[i]->remove(j);
            }
        }
    }
}

// Computes the appropriate block size and number of blocks based on the number of threads required and the number of SMPs
void StableMatchingGPU::getBlockNumberAndDimension(int n_threads, int *block_size, int *n_blocks){
    if (n_threads/_n_SMP >= 32){ //at least one warp per SMP
        *n_blocks = _n_SMP;
        *block_size = (n_threads + *n_blocks - 1) / *n_blocks;
    } else { //less than one warp per SMP
        *block_size = 32;
        *n_blocks = (n_threads + 31) / 32;
    }
}

// Repeats Fun2 until new_stack_mod_min_men is empty
void StableMatchingGPU::iterateFun2(){
    int n_threads, block_size, n_blocks;
    int *temp_p;
    n_threads = *_length_min_men_stack;
    getBlockNumberAndDimension(n_threads,&block_size,&n_blocks);

    apply_sm_constraint<<<n_blocks,block_size,0,_stream>>>(_n,_d_xpl,_d_ypl,_d_xPy,_d_yPx,_d_x_domain,_d_y_domain, _d_array_min_mod_men, _d_stack_mod_min_men, _d_length_min_men_stack, _d_new_stack_mod_min_men, _d_new_length_min_men_stack, _d_old_min_men, _d_max_men, _d_max_women);
    HANDLE_ERROR(hipMemcpyAsync(_new_length_min_men_stack, _d_new_length_min_men_stack, sizeof(int), hipMemcpyDeviceToHost, _stream));
    hipStreamSynchronize(_stream);
    
    while(*_new_length_min_men_stack!=0){
        HANDLE_ERROR(hipMemsetAsync(_d_array_min_mod_men,0,sizeof(int)*_n, _stream));
        *_length_min_men_stack = *_new_length_min_men_stack;
        *_new_length_min_men_stack = 0;
        HANDLE_ERROR(hipMemcpyAsync(_d_length_min_men_stack, _length_min_men_stack, sizeof(int) * 2, hipMemcpyHostToDevice, _stream));
        temp_p = _d_new_stack_mod_min_men;
        _d_new_stack_mod_min_men = _d_stack_mod_min_men;
        _d_stack_mod_min_men = temp_p;
        n_threads = *_length_min_men_stack;
        getBlockNumberAndDimension(n_threads,&block_size,&n_blocks);
        apply_sm_constraint<<<n_blocks,block_size,0,_stream>>>(_n,_d_xpl,_d_ypl,_d_xPy,_d_yPx,_d_x_domain,_d_y_domain, _d_array_min_mod_men, _d_stack_mod_min_men, _d_length_min_men_stack, _d_new_stack_mod_min_men, _d_new_length_min_men_stack, _d_old_min_men, _d_max_men, _d_max_women);
        HANDLE_ERROR(hipMemcpyAsync(_new_length_min_men_stack, _d_new_length_min_men_stack, sizeof(int), hipMemcpyDeviceToHost, _stream));
        hipStreamSynchronize(_stream);
    }
}

__device__ int getBitCuda(uint32_t* bitmap, int index){
    int offset = index % 32;
    return (bitmap[index/32] << offset) >> (sizeof (uint32_t)*8 - 1);
}

__device__ int getDomainBitCuda(uint32_t* bitmap, int row, int column, int n){
    return getBitCuda(bitmap,row*n+column);
}

__device__ void delBitCuda(uint32_t* bitmap, int index){
    int offset = index % 32;
    if ((bitmap[index>>5] << offset) >> (sizeof (uint32_t)*8 - 1) != 0){//index>>5 == index/32
        //bitwise and not
        atomicAnd(&bitmap[index>>5],~((UNS_ONE<< (sizeof (uint32_t)*8 - 1)) >> offset));//index>>5 == index/32
    }
}

__device__ void delDomainBitCuda(uint32_t* bitmap, int row, int column, int n){
    delBitCuda(bitmap,row*n+column);
}

__constant__ uint32_t ALL_ONES = 4294967295;

// f1: removes from the women's domains the men who don't have that woman in their list (domain) anymore, and vice versa
// Modifies only the domains
__global__ void make_domains_coherent(int n, int* xpl, int* ypl, int* xPy, int* yPx, uint32_t* x_domain, uint32_t* y_domain, int* stack_mod_men, int* stack_mod_women, int length_men_stack, int length_women_stack, int* stack_mod_min_men, int* length_min_men_stack, int* old_min_men, int* old_max_men, int* old_min_women, int* old_max_women){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    //closes redundant threads
    if (id>= length_men_stack + length_women_stack){
        return;
    }
    //gets person associated with thread and picks the correct data structures
    int person, other_person, other_index, temp;
    int is_man = id < length_men_stack;
    int *old_min, *old_max, *other_zPz, *zpl;
    uint32_t *person_domain, *other_domain;
    if(is_man){
        person = stack_mod_men[id];
        old_min =  old_min_men;
        old_max = old_max_men;
        person_domain = x_domain;
        other_domain = y_domain;
        zpl = xpl;
        other_zPz = yPx;
    } else {
        person = stack_mod_women[id - length_men_stack];
        old_min = old_min_women;
        old_max = old_max_women;
        person_domain = y_domain;
        other_domain = x_domain;
        zpl = ypl;
        other_zPz = xPy;
    }

    //scans the domain, looking for removed values
    for(int i=old_min[person]; i<=old_max[person];i++){
        if(getDomainBitCuda(person_domain,person,i,n)==0){//this bit is 0
            other_person = zpl[person*n+i];
            if(getDomainBitCuda(other_domain,other_person,other_zPz[other_person*n+person],n)){//==1 other person's domain must be updated
                other_index = other_zPz[other_person*n+person];
                delDomainBitCuda(other_domain,other_person,other_index,n);
                if(!is_man && old_min_men[other_person]==other_index){//updates stack_mod_min_men if other_person is a man and the min was just removed
                    temp = atomicAdd(length_min_men_stack,1);
                    stack_mod_min_men[temp]=other_person;
                }
            }
        }
    }
}

// f2: applies the stable marriage constraint
// Modifies old_min_men, max_women and x_domain
__global__ void apply_sm_constraint(int n, int* xpl, int* ypl, int* xPy, int* yPx, uint32_t* x_domain, uint32_t* y_domain, int* array_min_mod_men, int* stack_mod_min_men, int* length_min_men_stack, int* new_stack_mod_min_men, int* new_length_min_men_stack, int* old_min_men, int* max_men, int* max_women){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    //closes redundant threads
    if (id>= *length_min_men_stack){
        return;
    }

    //finds man assigned to this thread
    int m = stack_mod_min_men[id];

    //the variables named *_val represent the value of some person in the domain of another specific person of the opposite sex
    int w_index, w;
    int p_val, m_val;
    int succ_val, succ;
    int m_ith, w_val;

    //the thread cycles as long as it has a man assigned to it
    while(1){
        //finds the first woman remaining in m's domain/list
        w_index = old_min_men[m];
        if(w_index>max_men[m]){//empty domain
            old_min_men[m]=n;
            return;
        }else if(getDomainBitCuda(x_domain,m,w_index,n)){//value in domain
            w = xpl[m*n+w_index];

            m_val = yPx[w*n+m];

            //atomic read-and-write of max_women[w]
            p_val = atomicMin(max_women+w, m_val);

            if(m_val > p_val){//w prefers p to m
                old_min_men[m]=w_index+1; //atomicMax could be used, but it would very rarely make a difference
                //continue;//continues with the same m
            } else if(p_val==m_val){//w is already with m
                return;//the thread has no free man to find a woman for
            } else {//m_val<p, that is w prefers m to p
                succ_val = m_val + 1;
                while(succ_val<=p_val){
                    succ = ypl[w*n+succ_val];
                    delDomainBitCuda(x_domain,succ,xPy[succ*n+w],n);
                    succ_val++;
                }
                m = ypl[w*n+p_val];
                //continue;//continues with m:=p
            }
        }else{//value not in domain
            old_min_men[m]=w_index+1; //atomicMax could be used, but it would very rarely make a difference
            w = xpl[m*n+w_index];
            m_val = yPx[w*n+m];
            //atomic read-and-write of max_women[w]
            p_val = atomicMin(max_women+w, m_val-1);
            for(int i = m_val+1; i<=p_val; i++){//remove that woman from all the men that were removed from her domain (no need for m_val since the domains are coherent)
                if(getDomainBitCuda(y_domain,w,i,n)){//value wasn't already removed
                    m_ith=  ypl[w*n+i];
                    w_val = xPy[m_ith*n+w];
                    delDomainBitCuda(x_domain,m_ith,w_val,n);
                }
            }
            if(p_val>m_val-1){//checks if the min of the last man has changed (the condition checks if the max of the woman changed)
                m_ith=  ypl[w*n+p_val]; //necessary if a domain is empty
                w_val = xPy[m_ith*n+w]; //necessary if a domain is empty

                //marks the man as needing to be updated
                if(!atomicExch(&(array_min_mod_men[m_ith]),1)){ //atomic exchange to avoid duplicates (which could overflow the stack)
                    new_stack_mod_min_men[atomicAdd(new_length_min_men_stack,1)]=m_ith; //adds man to new stack
                }

            }
        }
        
    }
}

//f3: finalizes the changes in the domains and computes the new old_maxes and old_mins
// Modifies y_domain, old_max_women, old_max_men and old_min_women
__global__ void finalize_changes(int n, uint32_t* x_domain, uint32_t* y_domain, int* old_min_men, int* old_max_men, int* old_min_women, int* old_max_women, int* max_men, int* min_women, int* max_women){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    //closes redundant threads
    if (id>= n){
        return;
    }

    //finalizes women's domains
    int domain_offset = n * id;
    int first_bit_index = max_women[id]+1 + domain_offset; //need to add offset to find the domain of current woman, not the first one
    int last_bit_index = old_max_women[id] + domain_offset;
    int span = last_bit_index - first_bit_index + 1;
    int domain_index, n_bits, leftover_bits_in_word, offset;
    
    while(span>0){
        if(first_bit_index << (sizeof (int)*8 - 5) != 0 || span < 32){ //first_bit_index%32!=0, the last part of a word OR the first part of a word (beginning and/or end of area of interest)
            domain_index = first_bit_index>>5; //first_bit_index/32
            offset = first_bit_index%32; //offset of the first bit in the word
            leftover_bits_in_word = 32-offset; //the remaining bits from first_bit_index to the end of the word
            n_bits = leftover_bits_in_word<span ? leftover_bits_in_word : span; //how many bits to put in this word
            atomicAnd(&y_domain[domain_index],~((ALL_ONES<< (sizeof (int)*8 - n_bits)) >> offset)); //atomically deletes the appropriate bits of the word
            span-=n_bits; //marks some bits as added
            first_bit_index+=n_bits; //new index for the first bit that still hasn't been updated
        }else{//span>32, whole word can be written
            domain_index = first_bit_index>>5; //first_bit_index/32
            y_domain[domain_index]=0; //deletes whole word
            span-=32; //marks some bits as added
            first_bit_index+=32; //new index for the first bit that still hasn't been updated
        }
    }

    //updates old_max_men, old_min_women, old_max_women
    old_max_women[id]=max_women[id];

    int new_m=max_men[id];//old_max_men
    if(old_min_men[id]<=max_men[id]){
        while(new_m>=0 && getDomainBitCuda(x_domain,id,new_m,n)==0){
            new_m--;
        }
    }
    old_max_men[id]=new_m;

    new_m=min_women[id];//old_min_women
    if(max_women[id]>=min_women[id]){
        while(new_m<n && getDomainBitCuda(y_domain,id,new_m,n)==0){
            new_m++;
        }
    }
    old_min_women[id]=new_m;
}